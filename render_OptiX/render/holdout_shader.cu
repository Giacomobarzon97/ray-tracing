#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "Directional.h"
#include "structs.h"
#include "sampler.h"

using namespace optix;

//#define DIRECT
#define INDIRECT

// Standard ray variables
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// Variables for shading
rtBuffer<Directional> lights;
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

// Material properties (corresponding to OBJ mtl params)
rtDeclareVariable(float3, Ka, , );

// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );

// Recursive ray tracing variables
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(int, max_depth, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for Lambertian shading using the basic light as a directional source.
// This one includes shadows.
RT_PROGRAM void holdout_shader() 
{ 
  if(prd_radiance.depth > max_depth)
  {
    prd_radiance.result = make_float3(0.0f);
    return;
  }

  float3 hit_pos = ray.origin + t_hit*ray.direction;
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 
  float3 ffnormal = faceforward(normal, -ray.direction, normal); 

  Ray new_ray(hit_pos, ray.direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
  PerRayData_radiance prd_env;
  prd_env.emit = 1;
  prd_env.depth = prd_radiance.depth + 1;
  rtTrace(top_object, new_ray, prd_env);
  float3 rho_d = prd_env.result;

  float3 color = prd_radiance.emit ? Ka : make_float3(0.0f);
#ifdef DIRECT
  // Cast shadows due to direct illumination
  color += rho_d;
#endif
#ifdef INDIRECT
  // Ambient occlusion due to indirect illumination
  uint& t = prd_radiance.seed;

  color += rho_d;
#endif
  prd_radiance.result = color;
#if defined(DIRECT) && defined(INDIRECT)
  prd_radiance.result *= 0.5f;
#endif
}
