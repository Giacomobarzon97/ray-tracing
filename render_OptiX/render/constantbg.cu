#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "structs.h"

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, bg_color, , );

// Miss program returning background color
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}
