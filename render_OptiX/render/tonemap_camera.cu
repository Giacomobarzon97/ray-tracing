#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>

using namespace optix;

// Window variables
rtBuffer<float4, 2> output_buffer;
rtBuffer<float4, 2> tonemap_output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void tonemap_camera()
{
  float4 c = output_buffer[launch_index];
  tonemap_output_buffer[launch_index] = make_float4(powf(c.x, 1.0f/1.8f),
                                                    powf(c.y, 1.0f/1.8f),
                                                    powf(c.z, 1.0f/1.8f), c.w);
}
