#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../cuda/helpers.h"
#include "../cuda/random.h"
#include "structs.h"

using namespace optix;

// Camera variables
rtDeclareVariable(float3,   eye, , );
rtDeclareVariable(float3,   U, , );
rtDeclareVariable(float3,   V, , );
rtDeclareVariable(float3,   W, , );

// Ray generation variables
rtDeclareVariable(float,    scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint,     radiance_ray_type, , );
rtDeclareVariable(uint,     frame, , );

// Window variables
rtBuffer<float4, 2> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

// Exception and debugging variables
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW

RT_PROGRAM void pt_pinhole_camera()
{
#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif
  PerRayData_radiance prd;
   prd.emit = 1;
  prd.depth = 0;
  prd.seed = tea<16>(launch_dim.x*launch_index.y+launch_index.x, frame);
  prd.colorband = -1;
  
  float2 jitter = make_float2(rnd(prd.seed), rnd(prd.seed));
  float2 ip_coords = (make_float2(launch_index) + jitter) / make_float2(launch_dim) * 2.0f - 1.0f;
  float3 origin = eye;
  float3 direction = normalize(ip_coords.x*U + ip_coords.y*V + W);
  Ray ray(origin, direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  rtTrace(top_object, ray, prd);

#ifdef TIME_VIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  output_buffer[launch_index] = make_color( make_float3(  pixel_time ) ); 
#else
  float4 curr_sum = (frame != 0) ? output_buffer[launch_index] * ((float)frame) : make_float4(0.0f);
  output_buffer[launch_index] = (make_float4(prd.result, 0.0f) + curr_sum) / ((float)(frame + 1)) ;
#endif
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_float4(bad_color,1.0f);
}
