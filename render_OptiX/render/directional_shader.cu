#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "structs.h"
#include "Directional.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Variables for shading
rtBuffer<Directional> lights;
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

// Material properties (corresponding to OBJ mtl params)
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ka, , );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow() { rtTerminateRay(); }

// Closest hit program for Lambertian shading using the basic light as a directional source
RT_PROGRAM void directional_shader() 
{ 
  const float3& emission = Ka;
  const float3& rho_d = Kd;
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 
  float3 ffnormal = faceforward(normal, -ray.direction, normal); 

  // Implement Lambertian reflection here.
  //
  // Input:  
  // ray           (the ray that hit the material)
  //
  // Output: 
  // prd_radiance  (per ray data, in particular prd_radiance.result which is the reflected radiance)
  //
  // Relevant data fields that are available (see above):
  // lights        (vector of pointers to the lights in the scene)
  // ffnormal     (forward facing surface normal where the ray hit the material)
  // rho_d         (difuse reflectance of the material)
  //
  // Hint: (a) Elements in the lights vector have the data type Directional,
  //       which is defined in Directional.h.
  //       (b) OptiX includes a function dot(v, w) to take the dot product
  //       of two vectors v and w.
  float3 color=make_float3(0,0,0);
  for(int i=0;i<lights.size();i++){
	color=color+(rho_d/3.14)*lights[i].emission*dot(lights[i].direction,ffnormal);
  }

  prd_radiance.result = color + emission; 
}
