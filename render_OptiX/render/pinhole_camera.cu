#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "../cuda/helpers.h"
#include "../cuda/random.h"
#include "structs.h"

using namespace optix;

// Camera variables
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );

// Ray generation variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(uint,          frame, , );
rtBuffer<float2> jitter;

// Window variables
rtBuffer<float4, 2> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

// Exception and debugging variables
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW


RT_PROGRAM void pinhole_camera()
{
#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif
  float3 result = make_float3(0.0f); 
  for(int i = 0; i < jitter.size(); ++i)
  {
    float2 d = (make_float2(launch_index) + jitter[i]) / make_float2(launch_dim) * 2.0f - 1.0f;
    float3 ray_origin = eye;
    float3 ray_direction = normalize(d.x*U + d.y*V + W);
    
    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

    PerRayData_radiance prd;
  prd.emit = 1;
  prd.depth = 0;
  prd.seed = tea<16>(launch_dim.x*launch_index.y+launch_index.x, frame);
  prd.colorband = -1;
    rtTrace(top_object, ray, prd);
    result += prd.result;
  }
  result /= jitter.size();

#ifdef TIME_VIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  output_buffer[launch_index] = make_color( make_float3(  pixel_time ) ); 
#else
  output_buffer[launch_index] = make_float4( result, 0.0f );
#endif
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_float4( bad_color, 1.0f );
}
