#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "structs.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );

// Closest hit program for drawing shading normals
RT_PROGRAM void mirror_shader()
{
  if(prd_radiance.depth > max_depth)
  {
    prd_radiance.result = make_float3(0.0f);
    return;
  }

  float3 hit_pos = ray.origin + t_hit * ray.direction;
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

  // Implement mirror reflection here.
  //
  // Input:  
  // ray            (the ray that hit the material)
  // t_hit          (distance along ray to hit position)
  // prd_radiance   (per ray data)
  //
  // Output: 
  // prd_radiance   (per ray data)
  //
  // Relevant data fields that are available (see above):
  // shading_normal (surface normal where the ray hit the material)
  // max_depth      (maximum trace depth)
  // scene_epsilon  (user defined epsilon for ray tracing)
  // top_object     (topmost object in hierarchy of objects)
  //
  // Hint: (a) OptiX includes a function reflect(v, n) which returns the reflection
  //       of the ray direction v around the forward facing normal n.
  //       (b) Use prd_radiance.depth to keep track of the number of the number of
  //       surface interactions previously suffered by the ray.
  prd_radiance.result = make_float3(0.0f);
  if(prd_radiance.depth<=max_depth){
	  float3 reflected_dir=reflect(ray.direction,shading_normal);
	  PerRayData_radiance prd;
      float3 hit_pos=ray.direction*t_hit;
	  Ray r=make_Ray(hit_pos,reflected_dir,0,scene_epsilon,RT_DEFAULT_MAX);
	  rtTrace(top_object,r,prd_radiance);
	  prd_radiance.depth+=1;
  }
}
