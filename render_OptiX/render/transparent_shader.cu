#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "structs.h"
#include "fresnel.h"
#include "sampler.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(int, max_depth, , );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Material properties (corresponding to OBJ mtl params)
rtDeclareVariable(float, ior, , );

// Recursive ray tracing variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );

// Closest hit program for drawing shading normals
RT_PROGRAM void transparent_shader()
{
  if(prd_radiance.depth > max_depth)
  {
    prd_radiance.result = make_float3(0.0f);
    return;
  }

  float3 hit_pos = ray.origin + t_hit * ray.direction;
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 result = make_float3(0.0f);

  // Implement reflection and refraction using splitting.
  //
  // Input:  
  // ray            (the ray that hit the material)
  // t_hit          (distance along ray to hit position)
  // prd_radiance   (per ray data)
  //
  // Output: 
  // prd_radiance   (per ray data)
  //
  // Relevant data fields that are available (see above):
  // shading_normal (surface normal where the ray hit the material)
  // max_depth      (maximum trace depth)
  // scene_epsilon  (user defined epsilon for ray tracing)
  // top_object     (topmost object in hierarchy of objects)
  //
  // Hint: (a) OptiX includes functions reflect(...) and refract(...) for computing
  //       the reflected and refracted directions, respectively.
  //       (b) Make sure that you handle total internal reflection.

  // Trace reflected ray

  // Compute cosine to angle of incidence
  
  // Compute relative index of refraction

  // Compute Fresnel reflectance (R) and trace refracted ray if necessary
  result=make_float3(0.0f);
  PerRayData_radiance out_prd;
  out_prd.depth=prd_radiance.depth-1;
  if(prd_radiance.depth<=max_depth){
	  float3 reflected_dir=reflect(ray.direction,shading_normal);
	  PerRayData_radiance prd;
      float3 hit_pos=ray.direction*t_hit;
	  Ray reflect_r=make_Ray(hit_pos,reflected_dir,0,scene_epsilon,RT_DEFAULT_MAX);
	  rtTrace(top_object,reflect_r,prd_radiance);
	  prd_radiance.depth+=1;
     
      float3 refract_dir;
	  bool refraction=refract(refract_dir,ray.direction,shading_normal,ior);
	  Ray refract_r=make_Ray(hit_pos,refract_dir,0,scene_epsilon,RT_DEFAULT_MAX);
	  rtTrace(top_object,reflect_r,prd_radiance);

	  if(refraction){
		rtTrace(top_object,reflect_r,out_prd);
		out_prd.depth+=1;
	  }	  


	  float cos_theta=dot(ray.direction,shading_normal);
  }
  result=prd_radiance.result+out_prd.result;
  prd_radiance.result = result;
}
