#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "AreaLight.h"
#include "structs.h"
#include "sampler.h"

using namespace optix;

#define INDIRECT

// Standard ray variables
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

// Material properties (corresponding to OBJ mtl params)
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ka, , );

// Shadow variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );

#ifdef INDIRECT
// Recursive ray tracing variables
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(int, max_depth, , );
#endif

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for Lambertian shading using a triangle mesh as an area source.
// This one includes shadows.
RT_PROGRAM void arealight_shader() 
{
#ifdef INDIRECT
  if(prd_radiance.depth > max_depth)
  {
    prd_radiance.result = make_float3(0.0f);
    return;
  }
#endif
  float3 hit_pos = ray.origin + t_hit * ray.direction;
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 
  float3 ffnormal = faceforward(normal, -ray.direction, normal); 
  const float3& rho_d = Kd;
  uint& t = prd_radiance.seed;
  float3 color = prd_radiance.emit ? Ka : make_float3(0.0f);

  // Direct illumination
  float3 w_i;
  float3 L_e;
  float dist;
  sample_center(hit_pos, w_i, L_e, dist);

  // Implement Lambertian reflection here, include shadow rays.
  //
  // Input:  
  // ray           (the ray that hit the material)
  // t_hit         (distance along ray to hit position)
  //
  // Output: 
  // prd_radiance  (per ray data, in particular prd_radiance.result which is the reflected radiance)
  //
  // Relevant data fields that are available (see above):
  // w_i           (sampled direction toward the light)
  // L_e           (emitted radiance received from the direction w_i)
  // dist          (distance to the sampled position on the light source)
  // hit_pos       (position where the ray hit the material)
  // ff_normal     (forward facing surface normal where the ray hit the material)
  // rho_d         (difuse reflectance of the material)
  // scene_epsilon (user defined epsilon for ray tracing)
  // top_shadower  (topmost object in hierarchy of shadow casting objects)
  //
  // Hint: Implement the function sample_center(...) in AreaLight.h first.

  Ray r=make_Ray(hit_pos,w_i,0,scene_epsilon,t_hit-scene_epsilon);
  PerRayData_radiance prd;
  rtTrace(top_shadower,r,prd);
  color=color+(rho_d/3.14)*prd.result*dot(w_i,ffnormal);
  
  //color+=rho_d;
#ifdef INDIRECT
  // Indirect illumination
#endif

  prd_radiance.result = color; 
}
