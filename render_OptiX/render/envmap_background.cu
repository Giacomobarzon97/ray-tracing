#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011


#include <optix_world.h>
#include "structs.h"
#include "envmap.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Variables for shading
rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Miss program returning background color
RT_PROGRAM void miss()
{
  if(prd_radiance.emit)
    prd_radiance.result = env_lookup(ray.direction);
  else
    prd_radiance.result = make_float3(0.0f);
}
