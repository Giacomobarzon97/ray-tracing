#include "hip/hip_runtime.h"
// 02576 OptiX Rendering Framework
// Written by Jeppe Revall Frisvad, 2011
// Copyright (c) DTU Informatics 2011

#include <optix_world.h>
#include "structs.h"

using namespace optix;

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Variables for shading
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Any hit program for shadows
RT_PROGRAM void any_hit_shadow()
{
  prd_shadow.attenuation = 0.0f;
  rtTerminateRay();
}

// Closest hit program for drawing shading normals
RT_PROGRAM void normal_shader()
{
  float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  prd_radiance.result = normal*0.5f + 0.5f;
}