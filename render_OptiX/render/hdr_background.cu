#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "PerRayData.h"

// Standard ray variables
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Variables for shading
rtDeclareVariable(float3, bg_color, , );
rtTextureSampler<float4, 2> hdr_map; 

// Miss program returning background color
RT_PROGRAM void miss()
{
  float theta = atan2f( ray.direction.x, ray.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  prd_radiance.result = make_float3( tex2D(hdr_map, u, v) );
}
